
#include <hip/hip_runtime.h>
/** ExaTN: Tensor Runtime: Tensor network executor: NVIDIA cuQuantum
REVISION: 2021/12/30

Copyright (C) 2018-2021 Dmitry Lyakh
Copyright (C) 2018-2021 Oak Ridge National Laboratory (UT-Battelle)

Rationale:

**/

#ifdef CUQUANTUM

#include <cutensornet.h>
#include <cutensor.h>
#include <cuda_runtime.h>

#include <vector>
#include <unordered_map>
#include <type_traits>

#include <iostream>

#include "talshxx.hpp"

#include "cuquantum_executor.hpp"


#define HANDLE_CUDA_ERROR(x) \
{ const auto err = x; \
  if( err != cudaSuccess ) \
{ printf("Error: %s in line %d\n", cudaGetErrorString(err), __LINE__); std::abort(); } \
};

#define HANDLE_CTN_ERROR(x) \
{ const auto err = x; \
  if( err != CUTENSORNET_STATUS_SUCCESS ) \
{ printf("Error: %s in line %d\n", cutensornetGetErrorString(err), __LINE__); std::abort(); } \
};


namespace exatn {
namespace runtime {

struct TensorDescriptor {
 std::vector<int64_t> extents; //tensor dimension extents
 std::vector<int64_t> strides; //tensor dimension strides (optional)
 cudaDataType_t data_type;     //tensor element data type
 std::size_t volume = 0;       //tensor body volume
 std::size_t size = 0;         //tensor body size (bytes)
 void * src_ptr = nullptr;     //non-owning pointer to the tensor body source image
 std::vector<void*> dst_ptr;   //non-owning pointer to the tensor body dest image (for all GPU)
};

struct TensorNetworkReq {
 TensorNetworkQueue::ExecStat exec_status = TensorNetworkQueue::ExecStat::None; //tensor network execution status
 std::shared_ptr<numerics::TensorNetwork> network; //tensor network specification
 std::unordered_map<numerics::TensorHashType,TensorDescriptor> tensor_descriptors; //tensor descriptors (shape, volume, data type, body)
 std::unordered_map<unsigned int, std::vector<int32_t>> tensor_modes; //indices associated with tensor dimensions (key is the original tensor id)
 std::unordered_map<int32_t,int64_t> mode_extents; //extent of each registered tensor mode
 int32_t * num_modes_in = nullptr;
 int64_t ** extents_in = nullptr;
 int64_t ** strides_in = nullptr;
 int32_t ** modes_in = nullptr;
 uint32_t * alignments_in = nullptr;
 int32_t num_modes_out;
 int64_t * extents_out = nullptr;
 int64_t * strides_out = nullptr;
 int32_t * modes_out = nullptr;
 uint32_t alignment_out;
 std::vector<void*> memory_window_ptr; //end of the GPU memory segment allocated for the tensors
 cutensornetNetworkDescriptor_t net_descriptor;
 cutensornetContractionOptimizerConfig_t opt_config;
 cutensornetContractionOptimizerInfo_t opt_info;
 cutensornetContractionPlan_t comp_plan;
 cudaDataType_t data_type;
 cutensornetComputeType_t compute_type;
 cudaStream_t stream;
};


CuQuantumExecutor::CuQuantumExecutor(TensorImplFunc tensor_data_access_func):
 tensor_data_access_func_(std::move(tensor_data_access_func))
{
 static_assert(std::is_same<cutensornetHandle_t,void*>::value,"#FATAL(exatn::runtime::CuQuantumExecutor): cutensornetHandle_t != (void*)");

 const size_t version = cutensornetGetVersion();
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): cuTensorNet backend version " << version << std::endl;

 int num_gpus = 0;
 auto error_code = talshDeviceCount(DEV_NVIDIA_GPU,&num_gpus); assert(error_code == TALSH_SUCCESS);
 for(int i = 0; i < num_gpus; ++i){
  if(talshDeviceState(i,DEV_NVIDIA_GPU) >= DEV_ON){
   gpu_attr_.emplace_back(std::make_pair(i,DeviceAttr{}));
   gpu_attr_.back().second.workspace_ptr = talsh::getDeviceBufferBasePtr(DEV_NVIDIA_GPU,i);
   assert(reinterpret_cast<std::size_t>(gpu_attr_.back().second.workspace_ptr) % MEM_ALIGNMENT == 0);
   gpu_attr_.back().second.buffer_size = talsh::getDeviceMaxBufferSize(DEV_NVIDIA_GPU,i);
   std::size_t wrk_size = (std::size_t)(static_cast<float>(gpu_attr_.back().second.buffer_size) * WORKSPACE_FRACTION);
   wrk_size -= wrk_size % MEM_ALIGNMENT;
   gpu_attr_.back().second.workspace_size = wrk_size;
   gpu_attr_.back().second.buffer_size -= wrk_size;
   gpu_attr_.back().second.buffer_size -= gpu_attr_.back().second.buffer_size % MEM_ALIGNMENT;
   gpu_attr_.back().second.buffer_ptr = (void*)(((char*)(gpu_attr_.back().second.workspace_ptr)) + wrk_size);
   mem_pool_.emplace_back(LinearMemoryPool(gpu_attr_.back().second.buffer_ptr,
                                           gpu_attr_.back().second.buffer_size,MEM_ALIGNMENT));
  }
 }
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): Number of available GPUs = " << gpu_attr_.size() << std::endl;

 for(const auto & gpu: gpu_attr_){
  HANDLE_CUDA_ERROR(cudaSetDevice(gpu.first));
  HANDLE_CTN_ERROR(cutensornetCreate((cutensornetHandle_t*)(&gpu.second.cutn_handle)));
 }
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): Created cuTensorNet contexts for all available GPUs" << std::endl;

 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): GPU configuration:\n";
 for(const auto & gpu: gpu_attr_){
  std::cout << " GPU #" << gpu.first
            << ": wrk_ptr = " << gpu.second.workspace_ptr
            << ", size = " << gpu.second.workspace_size
            << "; buf_ptr = " << gpu.second.buffer_ptr
            << ", size = " << gpu.second.buffer_size << std::endl;
 }
}


CuQuantumExecutor::~CuQuantumExecutor()
{
 sync();
 for(const auto & gpu: gpu_attr_){
  HANDLE_CUDA_ERROR(cudaSetDevice(gpu.first));
  HANDLE_CTN_ERROR(cutensornetDestroy((cutensornetHandle_t)(gpu.second.cutn_handle)));
 }
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): Destroyed cuTensorNet contexts for all available GPUs" << std::endl;
 gpu_attr_.clear();
}


TensorNetworkQueue::ExecStat CuQuantumExecutor::execute(std::shared_ptr<numerics::TensorNetwork> network,
                                                        const TensorOpExecHandle exec_handle)
{
 assert(network);
 TensorNetworkQueue::ExecStat exec_stat = TensorNetworkQueue::ExecStat::None;
 auto res = active_networks_.emplace(std::make_pair(exec_handle, new TensorNetworkReq{}));
 if(res.second){
  auto tn_req = res.first->second;
  tn_req->network = network;
  tn_req->exec_status = TensorNetworkQueue::ExecStat::Idle;
  parseTensorNetwork(tn_req); //still Idle
  loadTensors(tn_req); //Idle --> Loading
  if(tn_req->exec_status == TensorNetworkQueue::ExecStat::Loading){
   planExecution(tn_req); //Loading --> Planning (while loading data)
   if(tn_req->exec_status == TensorNetworkQueue::ExecStat::Planning){
    contractTensorNetwork(tn_req); //Planning --> Executing
   }
  }
  exec_stat = tn_req->exec_status;
 }else{
  std::cout << "#WARNING(exatn::runtime::CuQuantumExecutor): execute: Repeated tensor network submission detected!\n";
 }
 return exec_stat;
}


TensorNetworkQueue::ExecStat CuQuantumExecutor::sync(const TensorOpExecHandle exec_handle,
                                                     int * error_code)
{
 *error_code = 0;
 TensorNetworkQueue::ExecStat exec_stat = TensorNetworkQueue::ExecStat::None;
 auto iter = active_networks_.find(exec_handle);
 if(iter != active_networks_.end()){
  auto tn_req = iter->second;
  if(tn_req->exec_status == TensorNetworkQueue::ExecStat::Executing){
   testCompletion(tn_req); //Executing --> Completed
  }else{
   if(tn_req->exec_status == TensorNetworkQueue::ExecStat::Idle)
    loadTensors(tn_req); //Idle --> Loading
   if(tn_req->exec_status == TensorNetworkQueue::ExecStat::Loading)
    planExecution(tn_req); //Loading --> Planning (while loading data)
   if(tn_req->exec_status == TensorNetworkQueue::ExecStat::Planning)
    contractTensorNetwork(tn_req); //Planning --> Executing
  }
  exec_stat = tn_req->exec_status;
  tn_req.reset();
  if(exec_stat == TensorNetworkQueue::ExecStat::Completed)
   active_networks_.erase(iter);
 }
 return exec_stat;
}


void CuQuantumExecutor::sync()
{
 while(!active_networks_.empty()){
  for(auto iter = active_networks_.begin(); iter != active_networks_.end(); ++iter){
   int error_code = 0;
   const auto exec_stat = sync(iter->first,&error_code); assert(error_code == 0);
   if(exec_stat == TensorNetworkQueue::ExecStat::Completed) break;
  }
 }
 return;
}


cudaDataType_t getCudaDataType(const TensorElementType elem_type)
{
 cudaDataType_t cuda_data_type;
 switch(elem_type){
 case TensorElementType::REAL32: cuda_data_type = CUDA_R_32F; break;
 case TensorElementType::REAL64: cuda_data_type = CUDA_R_64F; break;
 case TensorElementType::COMPLEX32: cuda_data_type = CUDA_C_32F; break;
 case TensorElementType::COMPLEX64: cuda_data_type = CUDA_C_64F; break;
 default:
  assert(false);
 }
 return cuda_data_type;
}


void CuQuantumExecutor::parseTensorNetwork(std::shared_ptr<TensorNetworkReq> tn_req)
{
 const auto & net = *(tn_req->network);
 const int32_t num_input_tensors = net.getNumTensors();
 tn_req->num_modes_in = new int32_t[num_input_tensors];
 tn_req->extents_in = new int64_t*[num_input_tensors];
 tn_req->strides_in = new int64_t*[num_input_tensors];
 tn_req->modes_in = new int32_t*[num_input_tensors];
 tn_req->alignments_in = new uint32_t[num_input_tensors];

 for(unsigned int i = 0; i < num_input_tensors; ++i) tn_req->strides_in[i] = NULL;
 for(unsigned int i = 0; i < num_input_tensors; ++i) tn_req->alignments_in[i] = MEM_ALIGNMENT;
 tn_req->strides_out = NULL;
 tn_req->alignment_out = MEM_ALIGNMENT;

 int32_t mode_id = 0, tens_num = 0;
 for(auto iter = net.cbegin(); iter != net.cend(); ++iter){
  const auto tens_id = iter->first;
  const auto & tens = iter->second;
  const auto tens_hash = tens.getTensor()->getTensorHash();
  const auto tens_vol = tens.getTensor()->getVolume();
  const auto tens_rank = tens.getRank();
  const auto tens_type = tens.getElementType();
  const auto & tens_legs = tens.getTensorLegs();
  const auto & tens_dims = tens.getDimExtents();

  auto res0 = tn_req->tensor_descriptors.emplace(std::make_pair(tens_hash,TensorDescriptor{}));
  if(res0.second){
   auto & descr = res0.first->second;
   descr.extents.resize(tens_rank);
   for(unsigned int i = 0; i < tens_rank; ++i) descr.extents[i] = tens_dims[i];
   descr.data_type = getCudaDataType(tens_type);
   descr.volume = tens_vol;
   descr.src_ptr = tensor_data_access_func_(*(tens.getTensor()),DEV_HOST,0,&(descr.size));
   assert(descr.src_ptr != nullptr);
  }

  auto res1 = tn_req->tensor_modes.emplace(std::make_pair(tens_id,std::vector<int32_t>(tens_rank)));
  assert(res1.second);
  for(unsigned int i = 0; i < tens_rank; ++i){
   const auto other_tens_id = tens_legs[i].getTensorId();
   const auto other_tens_leg_id = tens_legs[i].getDimensionId();
   auto other_tens_iter = tn_req->tensor_modes.find(other_tens_id);
   if(other_tens_iter == tn_req->tensor_modes.end()){
    res1.first->second[i] = ++mode_id;
    auto new_mode = tn_req->mode_extents.emplace(std::make_pair(mode_id,tens_dims[i]));
   }else{
    res1.first->second[i] = other_tens_iter->second[other_tens_leg_id];
   }
  }

  if(tens_id == 0){ //output tensor
   tn_req->num_modes_out = tens_rank;
   tn_req->extents_out = res0.first->second.extents.data();
   tn_req->modes_out = res1.first->second.data();
  }else{ //input tensors
   tn_req->num_modes_in[tens_num] = tens_rank;
   tn_req->extents_in[tens_num] = res0.first->second.extents.data();
   tn_req->modes_in[tens_num] = res1.first->second.data();
   ++tens_num;
  }
 }

 HANDLE_CTN_ERROR(cutensornetCreateNetworkDescriptor(gpu_attr_[0].second.cutn_handle,num_input_tensors,
                  tn_req->num_modes_in,tn_req->extents_in,tn_req->strides_in,tn_req->modes_in,tn_req->alignments_in,
                  tn_req->num_modes_out,tn_req->extents_out,tn_req->strides_out,tn_req->modes_out,tn_req->alignment_out,
                  tn_req->data_type,tn_req->compute_type,&(tn_req->net_descriptor)));

 HANDLE_CUDA_ERROR(cudaStreamCreate(&(tn_req->stream)));
 return;
}


void CuQuantumExecutor::loadTensors(std::shared_ptr<TensorNetworkReq> tn_req)
{
 
 return;
}


void CuQuantumExecutor::planExecution(std::shared_ptr<TensorNetworkReq> tn_req)
{
 
 return;
}


void CuQuantumExecutor::contractTensorNetwork(std::shared_ptr<TensorNetworkReq> tn_req)
{
 
 return;
}


void CuQuantumExecutor::testCompletion(std::shared_ptr<TensorNetworkReq> tn_req)
{
 
 return;
}

} //namespace runtime
} //namespace exatn

#endif //CUQUANTUM
